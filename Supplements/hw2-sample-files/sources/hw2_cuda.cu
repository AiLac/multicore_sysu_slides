#include "hip/hip_runtime.h"
#include <stdio.h>

#define MAX_BUFFER 10000

int height, width;
float data[MAX_BUFFER];

// 核函数统一使用该命名，参数列表可自定义
__global__ void kernel() 
{
}

int main(int argc, char *argv[])
{
	FILE *fp;
	fp = fopen(argv[1], "rb");
	fread(&height, sizeof(height), 1, fp);
	fread(&width, sizeof(width), 1, fp);
	fread(data, sizeof(float), height*width, fp);
	fclose(fp);

	//请编写计算过程
	// ...

	//以下是输出过程
	printf("%d %d\n",height,width);
	for(i=0;i<height;i++)
	{
		for(j=0;j<width;j++)
		{
			printf("%.02f ",result[i*width+j]);
		}
	}
	return 0;
}