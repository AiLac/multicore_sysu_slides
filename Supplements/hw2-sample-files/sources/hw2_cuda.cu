
#include <hip/hip_runtime.h>
#include <stdio.h>

#define MAX_BUFFER 10000

int height, width;
float data[MAX_BUFFER];

// 核函数统一使用该命名，参数列表可自定义
__global__ void kernel() 
{
}

int main(int argc, char *argv[])
{
	FILE *fp;
	fp = fopen(argv[1], "rb");
	fread(&height, sizeof(height), 1, fp);
	fread(&width, sizeof(width), 1, fp);
	fread(data, sizeof(float), height*width, fp);
	fclose(fp);
	return 0;
}